#include "hip/hip_runtime.h"
// A block can be split into parallel threads
// use parallel threads instead of parallel blocks
__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}


#define N 512
int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  // Alloc space for host copies of a, b, c and setup input values 
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size); 

  // Copy inputs to device 
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with Nthreads 
  add<<<1,N>>>(d_a, d_b, d_c);
  printf(a)
  printf(b)
  printf(c)

  // Copy results back to host 
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}



